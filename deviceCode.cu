#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "util.cuh"
#include "objects.cuh"
#include "reflectors.cuh"
#include <hiprand/hiprand_kernel.h>


/*__global__ void colorPixel (int w, int h, float4* colors)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= w || y >= h) return;

    int pixelIdx = y*w + x;

    //colors[pixelIdx] = make_float4(1.0f,1.0f,0.0f,0.0f);

    colors[pixelIdx] = make_float4 ((1.0f * x)/w,(1.0f * y)/w, 0.0f, 0.0f);
}*/
__device__ void triangleIntersect(Vertex* verts, Triangle* tri, const Ray& r, float4& barycentric, float& tval)
{
    float4 tria = verts[tri->aInd].position;
    float4 trib = verts[tri->bInd].position;
    float4 tric = verts[tri->cInd].position;
    float4 e1 = trib - tria;
    float4 e2 = tric - tria;

    float4 h = cross3(r.direction, e2);
    float a = dot(h, e1);

    if (fabs(a) < EPSILON)
    {
        barycentric = f4();
        tval = -1.0f;
        return;
    }
    float f = 1.0/a;

    float4 s = r.origin-tria;
    float u = f * dot(s, h);
    float4 q = cross3(s, e1);
    float v = f * dot(r.direction, q);
    float t = f * dot(e2, q);


    if (((u >= 0) && (v >= 0) && (u + v <= 1)) && t > EPSILON)
    {
        barycentric = f4(u, v, 1.0f-u-v);
        tval = t;
        return;
    }
    else
    {
        barycentric = f4();
        tval = -1.0f;
        return;
    }
}

__device__ bool aabbIntersect(const Ray& r, float4 minCorner, float4 maxCorner, float& tmin, float& tmax)
{
    tmin = -1e30f; // initialize to -infinity
    tmax = 1e30f;  // initialize to +infinity

    // Compute inverse ray direction once
    float4 invDir = make_float4(
        1.0f / r.direction.x,
        1.0f / r.direction.y,
        1.0f / r.direction.z,
        0.0f
    );

    // X axis
    float tx1 = (minCorner.x - r.origin.x) * invDir.x;
    float tx2 = (maxCorner.x - r.origin.x) * invDir.x;
    float tx_min = fminf(tx1, tx2);
    float tx_max = fmaxf(tx1, tx2);
    tmin = fmaxf(tmin, tx_min);
    tmax = fminf(tmax, tx_max);

    // Y axis
    float ty1 = (minCorner.y - r.origin.y) * invDir.y;
    float ty2 = (maxCorner.y - r.origin.y) * invDir.y;
    float ty_min = fminf(ty1, ty2);
    float ty_max = fmaxf(ty1, ty2);
    tmin = fmaxf(tmin, ty_min);
    tmax = fminf(tmax, ty_max);

    // Z axis
    float tz1 = (minCorner.z - r.origin.z) * invDir.z;
    float tz2 = (maxCorner.z - r.origin.z) * invDir.z;
    float tz_min = fminf(tz1, tz2);
    float tz_max = fmaxf(tz1, tz2);
    tmin = fmaxf(tmin, tz_min);
    tmax = fminf(tmax, tz_max);
    
    return (tmax >= tmin) && (tmax > 0.0f);
}

__device__ void BVHSceneIntersect(const Ray& r, BVHnode* BVH, int* BVHindices,Vertex* verts, Triangle* scene, Intersection& intersect, float max_t = 999999.0f, bool shortCircuit = false)
{
    intersect.valid = false;
    float min_t = 3.402823466e+38f;

    int nodeStack[64]; // Or 64. A stack depth of 32 is usually fine for a good BVH.
    int stackTop = 0;
    nodeStack[stackTop++] = 0; // Push the root node (index 0)

    while (stackTop > 0)
    {
        // Pop the next node to check
        int currentIndex = nodeStack[--stackTop];
        BVHnode& node = BVH[currentIndex];

        // 2. If it's a leaf node, check its triangles
        if (node.primCount > 0)
        {
            // This is your existing 'for' loop logic, which is correct.
            // Loop through primitives and update min_t and intersect.
            for (int i = node.first; i < node.primCount + node.first; i++)
            {
                int idx = BVHindices[i];
                Triangle* tri = &scene[idx];
                float4 barycentric;
                float t;
                triangleIntersect(verts, tri, r, barycentric, t);

                // NOTE: Here, max_t is your original max_t, 
                // but min_t is the *current closest hit*
                if (shortCircuit && ((t != -1.0f) && (t < min_t && t < max_t)))
                {
                    intersect.valid = true;
                    return;
                }
                else if ((t != -1.0f) && (t < min_t && t < max_t))
                {
                    min_t = t; // Update the closest-hit distance
                    intersect.point = r.at(t);
                    intersect.color = verts[tri->aInd].color * barycentric.z + verts[tri->bInd].color * barycentric.x + verts[tri->cInd].color * barycentric.y;
                    intersect.normal = normalize(verts[tri->aInd].normal * barycentric.z + verts[tri->bInd].normal * barycentric.x + verts[tri->cInd].normal * barycentric.y);
                    intersect.materialID = tri->materialID;
                    intersect.emission = tri->emission;
                    intersect.valid = true;
                    intersect.tri = *tri;
                }
            }
        }
        // 3. If it's an internal node, push its children onto the stack
        else
        {
            if (node.left >= 0 || node.right >= 0)
            {
                float tminL, tmaxL, tminR, tmaxR;
                bool hitLeft = false, hitRight = false;

                // Test left child if it exists
                if (node.left >= 0)
                    hitLeft = aabbIntersect(r, BVH[node.left].aabbMIN, BVH[node.left].aabbMAX, tminL, tmaxL);

                // Test right child if it exists
                if (node.right >= 0)
                    hitRight = aabbIntersect(r, BVH[node.right].aabbMIN, BVH[node.right].aabbMAX, tminR, tmaxR);

                // If both children were hit, push the farther one first
                if (hitLeft && hitRight)
                {
                    if (tminL < tminR)
                    {
                        nodeStack[stackTop++] = node.right; // farther
                        nodeStack[stackTop++] = node.left;  // nearer
                    }
                    else
                    {
                        nodeStack[stackTop++] = node.left;  // farther
                        nodeStack[stackTop++] = node.right; // nearer
                    }
                }
                else if (hitLeft)
                {
                    nodeStack[stackTop++] = node.left;
                }
                else if (hitRight)
                {
                    nodeStack[stackTop++] = node.right;
                }
            }
        }
    }
}

__device__ void sceneIntersection(const Ray& r, Vertex* verts, Triangle* scene, int triNum, 
    Intersection& intersect , float max_t = 999999.0f, bool shortCircuit = false)
{
    intersect.valid = false;
    float min_t = 3.402823466e+38f;
    
    for (int i = 0; i < triNum; i++)
    {
        Triangle* tri = &scene[i];
        float4 barycentric;
        float t;
        triangleIntersect(verts, tri, r, barycentric, t);
        if (shortCircuit && ((t != -1.0f) && (t < min_t && t < max_t)))
        {
            intersect.valid = true;
            return;
        }
        else if ((t != -1.0f) && (t < min_t && t < max_t))
        {
            min_t = t;
            intersect.point = r.at(t);
            //intersect.normal = verts[tri->aInd].normal;
            intersect.color = verts[tri->aInd].color * barycentric.z + verts[tri->bInd].color * barycentric.x + verts[tri->cInd].color * barycentric.y;
            intersect.normal = normalize(verts[tri->aInd].normal * barycentric.z + verts[tri->bInd].normal * barycentric.x + verts[tri->cInd].normal * barycentric.y);
            intersect.materialID = tri->materialID;
            intersect.emission = tri->emission;
            intersect.valid = true;
            intersect.tri = *tri;
        }
    }
}

__global__ void initRNG(hiprandState* states, int width, int height, unsigned long seed)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;
    hiprand_init(seed, idx, 0, &states[idx]);  
}

__device__ void nextEventEstimation(hiprandState& localState, BVHnode* BVH, int* BVHindices, const float4& wo, Vertex* vertices, int vertNum,
    Triangle* scene, int triNum, Triangle* lights, int lightNum, int materialID,const Intersection& intersect, 
    float& light_pdf, float4& contribution, Triangle* light = nullptr, const Intersection* newIntersect = nullptr)
{
    contribution = f4(0.0f,0.0f,0.0f);
    Triangle l;
    float4 apos;
    float4 bpos;
    float4 cpos;
    float u;
    float v;
    float4 p;
    float4 n;

    if (light == nullptr)
    {   
        if (lightNum == 0)
        {
            light_pdf = -1.0f;
            return;
        }
        int index = min(static_cast<int>(hiprand_uniform(&localState) * lightNum), lightNum - 1);
        l = lights[index];
        apos = vertices[l.aInd].position;
        bpos = vertices[l.bInd].position;
        cpos = vertices[l.cInd].position;
        u = sqrtf(hiprand_uniform(&localState));
        v = hiprand_uniform(&localState);

        p = (1.0f - u) * apos + u * (1.0f - v) * bpos + u * v * cpos; // point on light
        n = intersect.normal;
    }
    else 
    {
        l = *light;
        apos = vertices[l.aInd].position;
        bpos = vertices[l.bInd].position;
        cpos = vertices[l.cInd].position;
        p = newIntersect->point;
        n = newIntersect->normal;
    }
    


    float4 surfaceToLight = p-intersect.point;
    
    
    float4 wi = normalize(surfaceToLight);
    Ray r = Ray(intersect.point + n * EPSILON, wi);
    
    float t;
    float4 dummy;
    triangleIntersect(vertices, &l, r, dummy, t);
    
    Intersection sceneIntersect = Intersection();
    //sceneIntersection(r, vertices, scene, triNum, sceneIntersect, t*(0.9999), true);
    BVHSceneIntersect(r, BVH, BVHindices, vertices, scene, sceneIntersect, t*(0.9999), true);
    // following if statement tests for scene intersection (direct light) AND
    // whether the original light intersect was valid
    if (!sceneIntersect.valid && t != -1.0f) // direct LOS from intersection to light
    {
        float distanceSQR = lengthSquared(surfaceToLight);
        float4 lightNormal = vertices[l.aInd].normal;

        float cosThetaLight = fmaxf(dot(lightNormal, -wi), EPSILON);
        float cosThetaSurface = fmaxf(dot(n, wi), EPSILON);

        float G = cosThetaLight * cosThetaSurface/distanceSQR;
        float area = 0.5f * length(cross3(bpos - apos, cpos - apos));
        
        light_pdf = distanceSQR / (cosThetaLight * lightNum * area);
        float4 Le = l.emission;
        float4 f_val;
        cosine_f(intersect.color, f_val);

        contribution = f_val * Le / (light_pdf);
    }
    else {}
}

__global__ void Li (hiprandState* rngStates, BVHnode* BVH, int* BVHindices, int maxDepth, Vertex* vertices, int vertNum, Triangle* scene, int triNum, 
    Triangle* lights, int lightNum, int numSample, bool useMIS, int w, int h, float4* colors)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= w || y >= h) return;
    int pixelIdx = y*w + x;

    float4 colorSum = f4();

    hiprandState localState = rngStates[pixelIdx];
    float du = hiprand_uniform(&localState);
    float dv = hiprand_uniform(&localState);

    float4 a = f4();
    float4 cameraOrigin = f4();
    Ray r = Ray();
    float4 beta = f4();
    float4 Li = f4();
    int materialID = -1;
    int depth = -1;
    float4 f_val = f4();
    float4 wo_local = f4();
    float pdf = EPSILON;
    float4 wo_world = f4();
    float4 wi_world = f4();
    float4 wi_local = f4();
    float light_pdf = EPSILON;
    float4 nee = f4();
    Intersection intersect = Intersection();
    Intersection previousintersect = Intersection();
    float neeWeight = EPSILON; // reused for all MIS interactions
    float bsdfWeight = EPSILON; // reused for all MIS interactions
    float p = EPSILON;// for russian roulette
    float luminance = EPSILON; // for russian roulette

    for (int currSample = 0; currSample < numSample; currSample++)
    {   
        beta = f4(1.0f, 1.0f, 1.0f);
        Li = f4();

        du = hiprand_uniform(&localState);
        dv = hiprand_uniform(&localState);
        cameraOrigin = f4(0.0f,0.0f,1.0f);
        a = f4(cameraOrigin.x + (x + 1.0f*du - 0.5f - w/2.0f) * (1.0f / w), 
                        cameraOrigin.y + (y + 1.0f*dv - 0.5f - h/2.0f) * (1.0f / h),
                        cameraOrigin.z-1.0f);
        r.origin = cameraOrigin;
        r.direction = a-cameraOrigin;

        for (depth = 0; depth < 100; depth++)
        {
            intersect.valid = false;
            //sceneIntersection(r, vertices, scene, triNum, intersect);
            BVHSceneIntersect(r, BVH, BVHindices, vertices, scene, intersect);

            if (!intersect.valid) 
            {
                Li += beta * f4(0.4f,0.4f,0.7f);
                break;
            }
            materialID = intersect.materialID;

            if (lengthSquared(intersect.emission) > EPSILON)
            {
                if (depth == 0)
                {
                    Li += beta * intersect.emission;
                }
                else if (useMIS)
                {
                    nextEventEstimation(localState, BVH, BVHindices, wi_local, vertices, vertNum, scene, 
                        triNum, lights, lightNum, materialID, previousintersect, light_pdf, 
                        nee, &intersect.tri, &previousintersect);
                    
                    if (light_pdf > EPSILON)
                    {
                        bsdfWeight = pdf * pdf / (light_pdf * light_pdf 
                        + pdf * pdf);
                        Li += beta * intersect.emission * bsdfWeight;
                    }
                    
                    
                    
                }
            }

            toLocal(-r.direction, intersect.normal, wi_local);
            wi_world = normalize(-r.direction);

            if (useMIS)
            {
                nextEventEstimation(localState, BVH, BVHindices, wi_local, vertices, vertNum, scene, 
                triNum, lights, lightNum, materialID, intersect, light_pdf, nee);

                if (light_pdf > EPSILON)
                {
                    neeWeight = light_pdf * light_pdf / (fmaxf(wi_local.z, EPSILON)/PI * fmaxf(wi_local.z, EPSILON)/PI + light_pdf * light_pdf);

                    Li += beta * nee * neeWeight;
                }
                
            }

            cosine_f(intersect.color, f_val);
            cosine_sample_f(localState, wo_local, pdf);

            toWorld(wo_local, intersect.normal, wo_world);
            
            if (pdf < EPSILON) 
            break;

            beta *= (f_val * fabs(wo_local.z) / pdf);

            if (depth > maxDepth)
            {
                luminance = dot(beta, f4(0.2126f, 0.7152f, 0.0722f));
                p = clamp(luminance, 0.05f, 0.99f);

                if (hiprand_uniform(&localState) > p)   // survive with probability p
                    break;

                beta /= p;  // compensate for the survival probability
            }

            r.origin = intersect.point + intersect.normal * EPSILON;
            r.direction = wo_world;
            previousintersect = intersect;        
        }
        colorSum += Li;
    }
    colors[pixelIdx] = colorSum/numSample;
    rngStates[pixelIdx] = localState;
}

__host__ void launch(int maxDepth, BVHnode* BVH, int* BVHindices, Vertex* vertices, int vertNum, Triangle* scene, int triNum, 
    Triangle* lights, int lightNum, int numSample, bool useMIS, int w, int h, float4* colors)
{
    dim3 blockSize(16, 16);  
    dim3 gridSize((w+15)/16, (h+15)/16);
    hiprandState* d_rngStates;
    hipMalloc(&d_rngStates, w * h * sizeof(hiprandState));

    unsigned long seed = 103033UL;
    initRNG<<<gridSize, blockSize>>>(d_rngStates, w, h, seed);
    hipDeviceSynchronize();

    Li<<<gridSize, blockSize>>>(d_rngStates, BVH, BVHindices, maxDepth, vertices, vertNum, scene, triNum, 
        lights, lightNum, numSample, useMIS, w, h, colors);

    hipDeviceSynchronize();
    hipFree(d_rngStates);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error code: " << static_cast<int>(err) << std::endl;
        // only call this if the code isn't catastrophic
        if (err != hipErrorAssert && err != hipErrorUnknown)
            std::cerr << hipGetErrorString(err) << std::endl;
    }
    else
        std::cout << "no cuda error" << std::endl;
}